#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>
/*
PASOS SEGUIDOS PARA A�ADIR EL ARCHIVO DE CABECERAS: #include "common/book.h"
	- Clonar el repositorio de GitHub: https://github.com/CodedK/CUDA-by-Example-source-code-for-the-book-s-examples-
	- Ir a Proyect>Properties en VS
	- A�adir la raiz del repositorio clonado a: INCLUDE PATH
	- A�adir el directorio lib del repositorio clonado a: LIBRARY PATH
	- A�adir el directorio bin del repositorio clonado a: EXECUTABLE PATH
*/
#include "common/book.h"

#define TILE 8

template <class T>
__global__ void mulMatrixShared (T *Pd, T* Md, T* Nd, const int *width) {
	__shared__ T Mds[TILE][TILE];
	__shared__ T Nds[TILE][TILE];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by * TILE + ty;
	int col = bx * TILE + tx;
	float value = 0;
	for (int m = 0; m < *width / TILE; ++m) {
		Mds[ty][tx] = Md[row * *width + (m * TILE + tx)];
		Nds[ty][tx] = Nd[(m * TILE + ty) * *width + col];
		__syncthreads();
		for (int k = 0; k < TILE; ++k) {
			value += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	Pd[row * *width + col] = value;
}


template <class T>
std::string printMatrixOperation(const T *m1, const T *m2, const std::string &operation, const T *result, const int &dim) {
	int center_start = static_cast<int>(ceil(dim / 2)) - 1;
	int center_end = (center_start % 2 ? center_start + 1 : center_start) + 1;
	std::stringstream ss;
	auto matrixDrawer = [&](const int &init, const int &end, const std::string& c1, const std::string& c2) {
		for (int i = init; i < end; ++i) {
			std::stringstream ss_m1;
			std::stringstream ss_m2;
			std::stringstream ss_result;
			for (int j = 0; j < dim; ++j) {
				ss_m1 << m1[i + j * dim] << ", ";
				ss_m2 << m2[i + j * dim] << ", ";
				ss_result << result[i + j * dim] << ", ";
			}
			ss << ss_m1.str() << c1 << ss_m2.str() << c2 << ss_result.str() << "\n";
		}
	};
	matrixDrawer(0, center_start, "   ", "   ");
	matrixDrawer(center_start, center_end, " " + operation + " ", " = ");
	matrixDrawer(center_end, dim, "   ", "   ");
	return ss.str();
}

template <class T>
void fillVector(T *v, int size) {
	for (int i = 0; i < size; ++i) {
		v[i] = static_cast<int>(std::rand() % 9);
	}
}

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

int main() {
	int kwidth = 16*16;
	int kdim = static_cast<int>(sqrt(kwidth));
	dim3 dimBlock(TILE, TILE);
	dim3 dimGrid(16 / TILE, 16 / TILE);

	int *hm1 = (int *)malloc(sizeof(int)*kwidth);
	int *hm2 = (int *)malloc(sizeof(int)*kwidth);
	int *hresult = (int *)malloc(sizeof(int)*kwidth);
	int *dm1;
	int *dm2;
	int *dresult;
	int *ddim;

	fillVector<int>(hm1, kwidth);
	fillVector<int>(hm2, kwidth);

	HANDLE_ERROR(hipMalloc((void**)&dm1, sizeof(int)*kwidth));
	HANDLE_ERROR(hipMalloc((void**)&dm2, sizeof(int)*kwidth));
	HANDLE_ERROR(hipMalloc((void**)&dresult, sizeof(int)*kwidth));
	HANDLE_ERROR(hipMalloc((void**)&ddim, sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dm1, hm1, sizeof(int)*kwidth, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dm2, hm2, sizeof(int)*kwidth, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ddim, &kdim, sizeof(int), hipMemcpyHostToDevice));

	mulMatrixShared<int><<< dimGrid, dimBlock, 1 >>> (dresult, dm1, dm2, ddim);
	check_CUDA_Error("Kernell error");
	HANDLE_ERROR(hipMemcpy(hresult, dresult, sizeof(int)*kwidth, hipMemcpyDeviceToHost));
	std::cout << printMatrixOperation<int>(hm1, hm2, "*", hresult, kdim) << std::endl;

	free(hm1);
	free(hm2);
	free(hresult);
	HANDLE_ERROR(hipFree(dm1));
	HANDLE_ERROR(hipFree(dm2));
	HANDLE_ERROR(hipFree(ddim));
	HANDLE_ERROR(hipFree(dresult));

	getchar();
	return 0;
}
