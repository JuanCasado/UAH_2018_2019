#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>

#define MIN(a,b) (((a)<=(b))?(a):(b))

// N�mero de intervalos
const int N = 80000;
const int numThreads = 256;
const int bloquesPorGrid = MIN(32, (N + numThreads - 1) / numThreads);

__device__ float funcion_gpu(float x) {
	return (cos(1/x)+pow(x,2)*(pow(x,3)+9))/(x+3);
}
float funcion_cpu(float x) {
	return (cos(1 / x) + pow(x, 2)*(pow(x , 3) + 9)) / (x + 3);
}
__global__ void trapecios(float a, float b, float h, float* resultado) {
	__shared__ float parcial[numThreads];
	int iteracion = threadIdx.x + blockIdx.x* blockDim.x;
	float temp = 0;
	while (iteracion < N) {
		if (iteracion != 0) {
			temp += funcion_gpu(a + h * iteracion);
		}
		iteracion += blockDim.x * gridDim.x;
	}
	parcial[threadIdx.x] = temp; // Almacena los resultados parciales
	__syncthreads(); // Sincroniza threads
	int i = blockDim.x / 2;
	while (i != 0) { // Fase de reducci�n
		if (threadIdx.x < i) {
			parcial[threadIdx.x] += parcial[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}
	if (threadIdx.x == 0) {
		resultado[blockIdx.x] = parcial[0];
	}
}

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

int main(int argc, char **argv) {
	const float a = 1.5f;
	const float b = 2.78f;
	float h = (b - a) / N;
	float *resultado_cpu;
	float *resultado_gpu;
	resultado_cpu = (float*)malloc(bloquesPorGrid * sizeof(float));
	hipMalloc(&resultado_gpu, bloquesPorGrid * sizeof(float));
	check_CUDA_Error("RESERVA DE MEMORIA");
	trapecios << <bloquesPorGrid, numThreads >> > (a, b, h, resultado_gpu);
	check_CUDA_Error("LLAMADA AL KERNELL");
	hipMemcpy(resultado_cpu, resultado_gpu, bloquesPorGrid * sizeof(float), hipMemcpyDeviceToHost);
	check_CUDA_Error("COPIADO DE MEMORIA");
	float suma_parciales = (funcion_cpu(a) + funcion_cpu(b)) / 2.0f;
	for (int i = 0; i < bloquesPorGrid; i++) {
		suma_parciales += resultado_cpu[i];
	}
	suma_parciales *= h;
	std::cout << "Resultado de integral con GPU: " << suma_parciales << std::endl;
	suma_parciales = (funcion_cpu(a) + funcion_cpu(b)) / 2.0f;
	for (int i = 1; i < N; i++) {
		suma_parciales += funcion_cpu(a + i * h);
	}
	suma_parciales *= h;
	std::cout << "Resultado de integral con CPU: " << suma_parciales << std::endl;
	hipFree(resultado_gpu);
	check_CUDA_Error("LIBERACION DE MEMORIA");
	free(resultado_cpu);
	return 0;
}