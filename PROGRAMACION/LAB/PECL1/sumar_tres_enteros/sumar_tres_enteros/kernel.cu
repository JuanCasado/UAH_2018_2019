#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

template <class T>
__global__ T suma3 (const T a, const T b, const T c) {
	return a + b + c;
}

int main()
{
	int ha = 1;
	int hb = 2;
	int hc = 3;
	int da;
	int db;
	int dc;
	int hresult;
	int dresult;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

