#include <hip/hip_runtime.h>


#include <stdio.h>

/*
Adds three templated classes, they must implement the + operator.
Result is an I/O parameter.
*/
template <class T>
__global__ void add3 (T *result, const T *a, const T *b, const T *c) {
	*result =  *a + *b + *c;
}

int main() {
	int ha = 1;
	int hb = 2;
	int hc = 3;
	int *da;
	int *db;
	int *dc;
	int hresult;
	int *dresult;

	hipMalloc((void**)&da, sizeof(int));
	hipMalloc((void**)&db, sizeof(int));
	hipMalloc((void**)&dc, sizeof(int));
	hipMalloc((void**)&dresult, sizeof(int));

	hipMemcpy(da, &ha, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &hb, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dc, &hc, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dresult, &hresult, sizeof(int), hipMemcpyHostToDevice);

	add3<int><<<1, 1, 1>>>(dresult ,da, db, dc);

	hipMemcpy(&hresult, dresult, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d + %d + %d = %d\n", ha, hb, hc, hresult);

    return 0;
}

