#include <hip/hip_runtime.h>


#include <stdio.h>
#include <ctime>
#include <cstdlib>

/*
Adds three templated classes, they must implement the + operator.
Result is an I/O parameter.
*/
template <class T>
__global__ void add3 (T *result, const T *a, const T *b, const T *c) {
	*result =  *a + *b + *c;
}

int main() {
	std::srand(static_cast<int>(time(0)));
	int ha = static_cast<int>(std::rand() % 100 + 1);;
	int hb = static_cast<int>(std::rand() % 100 + 1);;
	int hc = static_cast<int>(std::rand() % 100 + 1);;
	int *da;
	int *db;
	int *dc;
	int hresult;
	int *dresult;

	//Allocate GPU memory.
	hipMalloc((void**)&da, sizeof(int));
	hipMalloc((void**)&db, sizeof(int));
	hipMalloc((void**)&dc, sizeof(int));
	hipMalloc((void**)&dresult, sizeof(int));

	//CPU --> GPU
	hipMemcpy(da, &ha, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &hb, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dc, &hc, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dresult, &hresult, sizeof(int), hipMemcpyHostToDevice);

	//Perform GPU operations
	add3<int><<<1, 1, 1>>>(dresult ,da, db, dc);

	//CPU <-- GPU
	hipMemcpy(&hresult, dresult, sizeof(int), hipMemcpyDeviceToHost);

	//Free GPU memory
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipFree(dresult);

	//Show result
	printf("%d + %d + %d = %d\n", ha, hb, hc, hresult);
    return 0;
}

