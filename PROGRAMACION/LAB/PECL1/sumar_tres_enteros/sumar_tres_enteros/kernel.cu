#include <hip/hip_runtime.h>


#include <stdio.h>

/*
Adds three templated classes, they must implement the + operator.
Result is an I/O parameter.
*/
template <class T>
__global__ void add3 (T *result, const T *a, const T *b, const T *c) {
	*result =  *a + *b + *c;
}

int main() {
	int ha = 1;
	int hb = 2;
	int hc = 3;
	int *da;
	int *db;
	int *dc;
	int hresult;
	int *dresult;

	//Allocate GPU memory.
	hipMalloc((void**)&da, sizeof(int));
	hipMalloc((void**)&db, sizeof(int));
	hipMalloc((void**)&dc, sizeof(int));
	hipMalloc((void**)&dresult, sizeof(int));

	//CPU --> GPU
	hipMemcpy(da, &ha, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, &hb, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dc, &hc, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dresult, &hresult, sizeof(int), hipMemcpyHostToDevice);

	//Perform GPU operations
	add3<int><<<1, 1, 1>>>(dresult ,da, db, dc);

	//CPU <-- GPU
	hipMemcpy(&hresult, dresult, sizeof(int), hipMemcpyDeviceToHost);

	//Show result
	printf("%d + %d + %d = %d\n", ha, hb, hc, hresult);
    return 0;
}

