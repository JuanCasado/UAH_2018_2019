
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

/*
PASOS SEGUIDOS PARA A�ADIR EL ARCHIVO DE CABECERAS: #include "common/book.h"
	- Clonar el repositorio de GitHub: https://github.com/CodedK/CUDA-by-Example-source-code-for-the-book-s-examples-
	- Ir a Proyect>Properties en VS
	- A�adir la raiz del repositorio clonado a: INCLUDE PATH
	- A�adir el directorio lib del repositorio clonado a: LIBRARY PATH
	- A�adir el directorio bin del repositorio clonado a: EXECUTABLE PATH
*/
#include "common/book.h"

int main() {
	int nDevices;
	HANDLE_ERROR(hipGetDeviceCount(&nDevices));
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf("\n------------ Device Number: %d ------------\n\n", i);
		printf("Device name: %s\n\n", prop.name);
		printf("Calculo como en el ejercicio de clase:\n");
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);

		printf("Total global memory %lld\n", prop.totalGlobalMem);
		printf("Total const memory %d\n", prop.totalConstMem);
		printf("Memory pitch %d\n\n", prop.memPitch);
		
		printf("Multiprocesor count %d\n", prop.multiProcessorCount);
		printf("Max Threads per multiprocesor %d\n", prop.maxThreadsPerMultiProcessor);
		printf("Shared memory per multiprocesor %d\n", prop.sharedMemPerMultiprocessor);
		printf("Registers per multiprocesor %d\n\n", prop.regsPerMultiprocessor);

		printf("Max Threads per block %d\n", prop.maxThreadsPerBlock);
		printf("Shared memory per block %d\n", prop.sharedMemPerBlock);
		printf("Registers per block %d\n", prop.regsPerBlock);
		printf("Max block size: ");
		printf("( [x->%d], [y->%d], [z->%d] )\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid size: ");
		printf("( [x->%d], [y->%d], [z->%d] )\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

		printf("Device Overlap %d\n", prop.deviceOverlap);
		printf("Kernel Execution Timeout %d\n", prop.kernelExecTimeoutEnabled);
	}
	getchar();
    return 0;
}