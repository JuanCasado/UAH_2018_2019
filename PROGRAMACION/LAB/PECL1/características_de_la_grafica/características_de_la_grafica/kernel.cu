
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

int main() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("\n------------ Device Number: %d ------------\n\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		
		printf("Max Threads per block %d\n", prop.maxThreadsPerBlock);
		printf("Shared memory per block %d\n\n", prop.sharedMemPerBlock);
		printf("Max block size: ");
		printf("(\n  [x->%d], \n  [y->%d], \n  [z->%d]\n)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid size: ");
		printf("(\n  [x->%d], \n  [y->%d], \n  [z->%d]\n)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	}
	getchar();
    return 0;
}