
#include "hip/hip_runtime.h"



#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>

template <class T>
__global__ void addVectors (T *result, T *v1, T *v2) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	result[i] = v1[i] + v2[i];
}

template <class T>
void fillVector(T *v, int size) {
	for (int i = 0; i < size; ++i) {
		v[i] = static_cast<int>(std::rand() % 20 + 1);
	}
}

template <class T>
std::string printOperation(const T *v1, const T *v2, const std::string &operation, const T *result, const int &size) {
	std::stringstream ss;
	for (int i = 0; i < size; ++i) {
		ss << v1[i] << " " << operation << " " << v2[i] << " = "<< result[i] << "\n";
	}
	return ss.str();
}


int main() {
	std::srand(static_cast<int>(time(0)));
	int *hv1;
	int *hv2;
	int *dv1;
	int *dv2;
	int *hresult;
	int *dresult;

	int split_factor = 4;
	int klenght = static_cast<int>(std::rand() % 10 + 1) * split_factor;
	int num_blocks = klenght / split_factor;
	int num_threads = split_factor;

	hv1 = static_cast<int *>(malloc(sizeof(int)*klenght));
	hv2 = static_cast<int *>(malloc(sizeof(int)*klenght));
	hresult = static_cast<int *>(malloc(sizeof(int)*klenght));
	fillVector<int>(hv1, klenght);
	fillVector<int>(hv2, klenght);

	hipMalloc((void**)&dv1, sizeof(int)*klenght);
	hipMalloc((void**)&dv2, sizeof(int)*klenght);
	hipMalloc((void**)&dresult, sizeof(int*)*klenght);

	hipMemcpy(dv1, hv1, sizeof(int)*klenght, hipMemcpyHostToDevice);
	hipMemcpy(dv2, hv2, sizeof(int)*klenght, hipMemcpyHostToDevice);

	addVectors<int> <<< num_blocks, num_threads, 1>>> (dresult, dv1, dv2);

	hipMemcpy(hresult, dresult, sizeof(int)*klenght, hipMemcpyDeviceToHost);

	std::cout << printOperation(hv1, hv2, "+", hresult, klenght) << std::endl;

	free(hv1);
	free(hv2);
	free(hresult);

	hipFree(dv1);
	hipFree(dv2);
	hipFree(dresult);

	getchar();
	return 0;
}