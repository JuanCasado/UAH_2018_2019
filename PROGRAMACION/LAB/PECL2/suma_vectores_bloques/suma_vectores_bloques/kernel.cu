
#include "hip/hip_runtime.h"



#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>

template <class T>
__global__ void addVectors (T *result, T *v1, T *v2) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	result[i] = v1[i] + v2[i];
}

template <class T>
void fillVector(T *v, int size) {
	for (int i = 0; i < size; ++i) {
		v[i] = static_cast<int>(std::rand() % 20 + 1);
	}
}

template <class T>
std::string printVector(const T *v, const int &size, const std::string &separator = ", ") {
	std::stringstream ss;
	ss << "{";
	for (int i = 0; i < size; ++i) {
		ss << v[i] << separator;
	}
	ss << "}";
	return ss.str();
}


int main() {
	std::srand(static_cast<int>(time(0)));
	int *hv1;
	int *hv2;
	int *dv1;
	int *dv2;
	int *hresult;
	int *dresult;

	//int klenght = static_cast<int>(std::rand() % 10 + 1);
	int klenght = 8;

	hv1 = static_cast<int *>(malloc(sizeof(int)*klenght));
	hv2 = static_cast<int *>(malloc(sizeof(int)*klenght));
	hresult = static_cast<int *>(malloc(sizeof(int)*klenght));
	fillVector<int>(hv1, klenght);
	fillVector<int>(hv2, klenght);

	hipMalloc((void**)&dv1, sizeof(int)*klenght);
	hipMalloc((void**)&dv2, sizeof(int)*klenght);
	hipMalloc((void**)&dresult, sizeof(int*)*klenght);

	hipMemcpy(dv1, hv1, sizeof(int)*klenght, hipMemcpyHostToDevice);
	hipMemcpy(dv2, hv2, sizeof(int)*klenght, hipMemcpyHostToDevice);

	int split_factor = 4;
	int num_blocks = klenght / split_factor;
	int num_threads = split_factor;
	addVectors<int> <<< num_blocks, num_threads, 1>>> (dresult, dv1, dv2);

	hipMemcpy(hresult, dresult, sizeof(int)*klenght, hipMemcpyDeviceToHost);

	std::cout << printVector<int>(hv1, klenght);
	std::cout << " + ";
	std::cout << printVector<int>(hv2, klenght);
	std::cout << " = ";
	std::cout << printVector<int>(hresult, klenght);
	std::cout << std::endl;

	free(hv1);
	free(hv2);
	free(hresult);

	hipFree(dv1);
	hipFree(dv2);
	hipFree(dresult);

	getchar();
	return 0;
}