#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>
/*
PASOS SEGUIDOS PARA A�ADIR EL ARCHIVO DE CABECERAS: #include "common/book.h"
	- Clonar el repositorio de GitHub: https://github.com/CodedK/CUDA-by-Example-source-code-for-the-book-s-examples-
	- Ir a Proyect>Properties en VS
	- A�adir la raiz del repositorio clonado a: INCLUDE PATH
	- A�adir el directorio lib del repositorio clonado a: LIBRARY PATH
	- A�adir el directorio bin del repositorio clonado a: EXECUTABLE PATH
*/
#include "common/book.h"

template <class T>
__global__ void mulMatrix(T *result, const T *m1, const T *m2, const int *dim, const int *tile) {
	int col = blockIdx.x * *tile + threadIdx.x;
	int row = blockIdx.y * *tile + threadIdx.y;
	T m_mul = 0;
	for (int i = 0; i < *dim; ++i) {
		m_mul += m1[row * *dim + i] * m2[col + *dim * i];
	}
	result[row * *dim + col] = m_mul;
}

template <class T>
std::string printMatrixOperation(const T *m1, const T *m2, const std::string &operation, const T *result, const int &dim) {
	int center_start = static_cast<int>(ceil(dim/2)) - 1;
	int center_end = (center_start % 2 ? center_start + 1: center_start) + 1;
	std::stringstream ss;
	auto matrixDrawer = [&](const int &init, const int &end, const std::string& c1, const std::string& c2) {
		for (int i = init; i < end; ++i) {
			std::stringstream ss_m1;
			std::stringstream ss_m2;
			std::stringstream ss_result;
			for (int j = 0; j < dim; ++j) {
				ss_m1 << m1[i+j*dim] << ", ";
				ss_m2 << m2[i+j*dim] << ", ";
				ss_result << result[i+j*dim] << ", ";
			}
			ss << ss_m1.str() << c1 << ss_m2.str() << c2 << ss_result.str() << "\n";
		}
	};
	matrixDrawer(0, center_start, "   ", "   ");
	matrixDrawer(center_start, center_end, " " +operation+ " ", " = ");
	matrixDrawer(center_end, dim, "   ", "   ");
	return ss.str();
}

template <class T>
void fillVector(T *v, int size) {
	for (int i = 0; i < size; ++i) {
		v[i] = static_cast<int>(std::rand() % 9);
	}
}

__host__ void check_CUDA_Error(const char *mensaje) {
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if (error != hipSuccess) {
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error), mensaje); printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla = getchar();
		exit(-1);
	}
}

int main(){
	std::srand(static_cast<int>(time(0)));
	int klenght = 16*16;
	int kdim = static_cast<int>(sqrt(klenght));
	int ktile = kdim/2;
	dim3 dimGrid(kdim / ktile, kdim / ktile);
	dim3 dimBlock(ktile, ktile);
	int *hm1 = (int *)malloc(sizeof(int)*klenght);
	int *hm2 = (int *)malloc(sizeof(int)*klenght);
	int *hresult = (int *)malloc(sizeof(int)*klenght);
	int *dm1;
	int *dm2;
	int *dresult;
	int *ddim;
	int *dtile;

	fillVector<int>(hm1, klenght);
	fillVector<int>(hm2, klenght);

	HANDLE_ERROR(hipMalloc((void**)&dm1, sizeof(int)*klenght));
	HANDLE_ERROR(hipMalloc((void**)&dm2, sizeof(int)*klenght));
	HANDLE_ERROR(hipMalloc((void**)&dresult, sizeof(int)*klenght));
	HANDLE_ERROR(hipMalloc((void**)&ddim, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dtile, sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dm1, hm1, sizeof(int)*klenght, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dm2, hm2, sizeof(int)*klenght, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ddim, &kdim, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dtile, &ktile, sizeof(int), hipMemcpyHostToDevice));

	mulMatrix<int> <<< dimGrid, dimBlock, 1 >>> (dresult, dm1, dm2, ddim, dtile);
	check_CUDA_Error("Kernell error");
	HANDLE_ERROR(hipMemcpy(hresult, dresult, sizeof(int)*klenght, hipMemcpyDeviceToHost));
	std::cout << printMatrixOperation<int>(hm1, hm2, "*", hresult, kdim) << std::endl;

	free(hm1);
	free(hm2);
	free(hresult);
	HANDLE_ERROR(hipFree(dm1));
	HANDLE_ERROR(hipFree(dm2));
	HANDLE_ERROR(hipFree(ddim));
	HANDLE_ERROR(hipFree(dtile));
	HANDLE_ERROR(hipFree(dresult));

	getchar();
    return 0;
}
