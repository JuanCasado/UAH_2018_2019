
#include "hip/hip_runtime.h"

#include <stdlib.h>


#include <stdio.h>
#include <iostream>

__global__ void set2 (int *dnum) {
	*dnum = 2;
}

int main() {
	int hnum = 0, *dnum;
    
	hipMalloc((void**) &dnum, sizeof(int));
	hipMemcpy(dnum, &hnum, sizeof(int), hipMemcpyHostToDevice);

	set2 <<< 1, 1 >>> (dnum);

	hipMemcpy(&hnum, dnum, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "a = " << hnum << std::endl;
	hipFree(dnum);

	getchar();
    return 0;
}

